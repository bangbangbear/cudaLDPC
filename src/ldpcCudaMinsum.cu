#include "hip/hip_runtime.h"
#include "ldpcCudaMinsum.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "device_functions_decls.h"

#define CIRC_SIZE 128

__global__ void update_v2c_kernel(float *v2c, float * c2v, int *hd, const float *llr, const ldpcMatrixQC::circ_entry *col_circ, const int *col_weight, int max_colWeight, int numCircRows, int numCircCols)
{
  const int col = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float sum_llr[CIRC_SIZE];
  const int weight = col_weight[blockIdx.x];
  const ldpcMatrixQC::circ_entry *this_col = col_circ + max_colWeight * blockIdx.x;
  int i = 0;

  sum_llr[threadIdx.x] = llr[col];
  for(i = 0; i < weight; i++) {
    int offset = (threadIdx.x - this_col[i].offset) & 0x7f;
    int circ_ind = this_col[i].ind * numCircCols + blockIdx.x;
    sum_llr[threadIdx.x] += c2v[circ_ind * blockDim.x + offset];
  }

  for(i = 0; i < weight; i++) {
    int offset = (threadIdx.x - this_col[i].offset) & 0x7f;
    int circ_ind = this_col[i].ind * numCircCols + blockIdx.x;
    v2c[circ_ind * blockDim.x + offset] = sum_llr[threadIdx.x] - c2v[circ_ind * blockDim.x + offset];
  }
  hd[col] = sum_llr[threadIdx.x] < 0;
}

__global__ void update_c2v_kernel(float *v2c, float *c2v, int *usc, ldpcMatrixQC::circ_entry *circ_row, int *row_weight, int max_colWeight, int numCircRows, int numCircCols)
{
  float min1 = 1e32, min2;
  int min_ind, sign = 1, i;
  int weight = row_weight[blockIdx.x];
  ldpcMatrixQC::circ_entry *this_row = circ_row + max_colWeight * blockIdx.x;

  if(blockIdx.x == 0 && threadIdx.x == 0) {
    usc[0] = 0;
  }

  for(i=0; i < weight; i++) {
    int col = this_row[i].ind;
    int circ_ind = blockIdx.x * numCircCols + col;
    float mag = v2c[circ_ind * blockDim.x + threadIdx.x];
    if(mag < 0) {
      sign = -sign;
      mag = -mag;
    }
    if(mag < min1) {
      min2 = min1;
      min1 = mag;
      min_ind = col;
    } else if (mag < min2) {
      min2 = mag;
    }
  }

  for(i = 0; i < weight; i++) {
    int col = this_row[i].ind;
    int circ_ind = blockIdx.x * numCircCols + col;
    c2v[circ_ind * blockDim.x + threadIdx.x] = (((sign * v2c[circ_ind * blockDim.x + threadIdx.x]) < 0) ? -0.75 : 0.75) * (min_ind == col ? min2 : min1);
  }

  if(sign < 0) usc[0] = 1; //just want to see if usc is non-zero, so no need to use atomicAdd
}


ldpcCudaMinsumDec::ldpcCudaMinsumDec(ldpcMatrixQC const &mat) :
  ldpcMinSumQCDec(mat)
{
  // prepare host data
  std::vector<std::vector<ldpcMatrixQC::circ_entry>> row_circs = mat.get_row_circulants();
  std::vector<int> row_weight = mat.get_row_weight();
  max_rowWeight = *std::max_element(row_weight.begin(), row_weight.end());
  for(auto it = row_circs.begin(); it != row_circs.end(); ++it) {
    for(int i = 0; i < max_rowWeight; i++) {
      if(i < (int)it->size()) {
        row_circs_host.push_back((*it)[i]);
      } else {
        ldpcMatrixQC::circ_entry e = {-1, -1};
        row_circs_host.push_back(e);
      }
    }
  }

  std::vector<std::vector<ldpcMatrixQC::circ_entry>> col_circs = mat.get_col_circulants();
  std::vector<int> col_weight = mat.get_col_weight();
  max_colWeight = *std::max_element(col_weight.begin(), col_weight.end());
  for(auto it = col_circs.begin(); it != col_circs.end(); ++it) {
    for(int i = 0; i < max_colWeight; i++) {
      if(i < (int)it->size()) {
        col_circs_host.push_back((*it)[i]);
      } else {
        ldpcMatrixQC::circ_entry e = {-1, -1};
        col_circs_host.push_back(e);
      }
    }
  }
  
  int device_id = 0; // Force to use my only GPU
  checkCudaErrors(hipSetDevice(device_id));
  checkCudaErrors(hipDeviceReset());
  checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

  // Allocate memory on GPU
  checkCudaErrors(hipMalloc((void **)&row_circs_cuda, row_circs_host.size() * sizeof(row_circs_host[0])));
  checkCudaErrors(hipMemcpy(row_circs_cuda, &row_circs_host[0], row_circs_host.size() * sizeof(row_circs_host[0]), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **)&col_circs_cuda, col_circs_host.size() * sizeof(col_circs_host[0])));
  checkCudaErrors(hipMemcpy(col_circs_cuda, &col_circs_host[0], col_circs_host.size() * sizeof(col_circs_host[0]), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **)&row_weight_cuda, row_weight.size() * sizeof(int)));
  checkCudaErrors(hipMemcpy(row_weight_cuda, &row_weight[0], row_weight.size() * sizeof(int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMalloc((void **)&col_weight_cuda, col_weight.size() * sizeof(int)));
  checkCudaErrors(hipMemcpy(col_weight_cuda, &col_weight[0], col_weight.size() * sizeof(int), hipMemcpyHostToDevice));

  // work space
  checkCudaErrors(hipMalloc((void **)&v2c_cuda, numCircRows * numCircCols * circSize * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&c2v_cuda, numCircRows * numCircCols * circSize * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&llr_cuda, numCols  * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&hd_cuda, numCols * sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&usc_cuda, sizeof(int)));
}

ldpcCudaMinsumDec::~ldpcCudaMinsumDec()
{
  checkCudaErrors(hipFree(v2c_cuda));
  checkCudaErrors(hipFree(c2v_cuda));
  checkCudaErrors(hipFree(llr_cuda));
  checkCudaErrors(hipFree(hd_cuda));

  checkCudaErrors(hipFree(row_weight_cuda));
  checkCudaErrors(hipFree(row_circs_cuda));
  checkCudaErrors(hipFree(col_weight_cuda));
  checkCudaErrors(hipFree(col_circs_cuda));

  checkCudaErrors(hipDeviceReset());
}

std::vector<int> ldpcCudaMinsumDec::decode(std::vector<float> const &llr)
{
  int usc_host = -1;
  std::vector<int> hd_host(numCols);
  checkCudaErrors(hipMemcpy(llr_cuda, &llr[0], llr.size() * sizeof(float), hipMemcpyHostToDevice));
  // checkCudaErrors(hipMemset(v2c_cuda, 0, numCircRows * numCircCols * CIRC_SIZE * sizeof(float)));
  checkCudaErrors(hipMemset(c2v_cuda, 0, numCircRows * numCircCols * CIRC_SIZE * sizeof(float)));

  for(int iter = 0; iter < maxIter && usc_host; iter++) {
    update_v2c_kernel<<<numCols/circSize, circSize>>>(v2c_cuda, c2v_cuda, hd_cuda, llr_cuda, col_circs_cuda, col_weight_cuda, max_colWeight, numCircRows, numCircCols);
    update_c2v_kernel<<<numRows/circSize, circSize>>>(v2c_cuda, c2v_cuda, usc_cuda, row_circs_cuda, row_weight_cuda, max_rowWeight, numCircRows, numCircCols);

    checkCudaErrors(hipMemcpy(&usc_host, usc_cuda, sizeof(int), hipMemcpyDeviceToHost));
  }

  checkCudaErrors(hipMemcpy(&hd_host[0], hd_cuda, numCols * sizeof(int), hipMemcpyDeviceToHost));


  return hd_host;
}
